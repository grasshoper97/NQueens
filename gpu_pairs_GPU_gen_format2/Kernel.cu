#include "hip/hip_runtime.h"
/*
* 2018.12.3  
* add global vars d_pairs_i, d_pairs_j, pair_size 
* h_result/d_result/reaturn type change to unsigned int  
* 20:00
* generate pairs[] form cpu to GPU.
*/
#include "Kernel.h"
#define DEBUG 1
#define BLOCK_SIZE 512 


//..................................................................
__global__ void Ker_Warm(){
	// empty body, just Ker_Warm GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

	
// tid : 1~ N(N-1)/2
__device__ void getPairs(
	int N, 						// Queen number;
	int pos, 	// position in pairs[];
	int *Xi, 					// return i;
	int *Xj)					// return j;
{

	int iRound=N-1;
	for(int i =0 ; i < N-1; i ++ ){ // N-1 round totally , 
		if(pos - iRound<= 0){
			//printf("\033[1;40;32m  (%d,%lld)\n  \033[0m",i, i+pos );
			*Xi=i;
			*Xj=i+pos;
			break;
		}
		pos -= iRound;
		iRound--;
	}
}

// tid : 0~ N(N-1)/2-1
__device__ void getPairs_4(
	int N, 						// Queen number;
	long long int tid, 	// position in pairs[];
	int *Xi, 					// return i;
	int *Xj)					// return j;
{

	int NumofRound = N-1;
	for(int iRound =0 ; iRound < N-1; iRound ++ ){ // N-1 round totally , 
		if(tid - NumofRound< 0){
			*Xi=iRound;
			*Xj= tid +1 +iRound;
			break;
		}
		tid -= NumofRound;
		NumofRound--;
	}
}

__global__ void Ker_Check_Combination (
		int *d_combination,		// a combination to be checkeda, store in d_com[1~N], d_com[0] is unused, for caculate gradient esaily.
		int combination_size,	// length of combinations =queen number =N
		unsigned long long int pairs_size,			// pair number , =N*(N-1)/2
		unsigned int *d_result
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// tid is int , max is 2^31-1, K80 max thread number.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid>=pairs_size)  // pairs_size = N*(N-1)/2.   tid in [0.. pairs_sizea] , tid +1 in [1.. pairs_size]
		return;


	// N*(N-1)/2 threads check pair
	int Xi;
	int Xj;
	/* getPairs( combination_size, tid+1, &Xi, &Xj);  // getPairs use [1.. pairs_size], tid start at 0. so tid+1 as no. */
	getPairs_4( combination_size, tid, &Xi, &Xj);  // getPairs use [1.. pairs_size], tid start at 0. so tid+1 as no.

	int Yi = d_combination[Xi];  // i,j in [1 ~~ N]
	int Yj = d_combination[Xj];
	if( Yi == Yj || Xi+Yi == Xj+Yj  ||  Yi -Xi == Yj - Xj) { //use +- instead of *,  can judge random sequence 
		//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", Xj,Yj,Xi,Yi,tid);
		atomicAdd ((unsigned int *)&d_result[0],1);
		//printf("add 1: block =%d, thread=%d, tid=%d pair (%2d,%2d)\n",blockIdx.x, threadIdx.x ,tid, Xi, Xj);
	}


}// end of Kernel

//............................ cpu Interface......................................


void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;

	hipEventRecord(start, 0);
		Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	checkCudaErrors(hipDeviceSynchronize());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","Ker_Warm() run time=",cuda_time);
	#endif
}

unsigned int get_conflicts(int * combination, int combination_size)
{
	int *h_combination = 0;  //store a number in [1~N]
	int *d_combination = 0;
	unsigned int *h_result= 0;
	unsigned int *d_result= 0;  // for automic opration
	hipError_t cuda_err; 
	h_result        =(unsigned int *)malloc(	1                * sizeof(unsigned int));

	if(  h_result==NULL )
	{
		printf("malloc h_result  error \ni");
		exit(1);
	}
	h_result[0]=0;
	h_combination= combination;
	int pairs_size= combination_size * ( combination_size -1) /2;

	cuda_err = hipSuccess;
	// allocate GPU mem
	checkCudaErrors(hipMalloc((void **)&d_combination, combination_size * sizeof(int)));  // if bigger than 4G, GPU memory alloc false;
	checkCudaErrors(hipMalloc((void **)&d_result ,     1                * sizeof(unsigned int)));
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "alloc d_combination error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		printf( "alloc d_combination error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//combination  H->D	
	checkCudaErrors(hipMemcpy(d_combination, h_combination , combination_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_result		, h_result		, 1 			   * sizeof(unsigned int), hipMemcpyHostToDevice)); 


	//call Kernel<<< >>>
	Ker_Check_Combination<<<   (pairs_size + BLOCK_SIZE-1)/BLOCK_SIZE ,  BLOCK_SIZE  >>>
				(d_combination ,combination_size  , pairs_size, d_result);  

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		if(DEBUG) fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
		if(DEBUG) fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));


	// D->H  and timer	
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(h_result, d_result, 1 * sizeof(int), hipMemcpyDeviceToHost)); // wait for Kernel finished.
	cuda_err = hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		if(DEBUG) fprintf(stderr, "D->H error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
		if(DEBUG) fprintf(stderr, "D->H successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	unsigned int conflicts = h_result[0];
	//************************************************************************************************************

	//free memory
	free(h_result);
	checkCudaErrors(hipFree(d_combination));
	checkCudaErrors(hipFree(d_result));

	return  conflicts;
}
