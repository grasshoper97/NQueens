#include "hip/hip_runtime.h"
 /*
  * 2019.4.15 复制自 SDK simpleMultiGPU， 简化头文件，用来尝试双卡并行。
  */
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


typedef struct
{
    //Host-side input data
    int dataN;
    float *h_Data;
    //Partial sum for this GPU
    float *h_Sum;

    //Device buffers
    float *d_Data,*d_Sum;

    //Reduction copied back from GPU
    float *h_Sum_from_device;

    //Stream for asynchronous command execution
    hipStream_t stream;

} TGPUplan;
////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
//const int DATA_N        = 1048576 * 32; // 32M
/* const int DATA_N        = 1048576 * 512; // 32M */

////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA Sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N)
{
    const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x; //lean: thread number total;
    float sum = 0;

    for (int pos = tid; pos < N; pos += threadN)
        sum += d_Input[pos];

    d_Result[tid] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    TGPUplan      plan[MAX_GPU_COUNT]; //对应每个GPU的结构体
    float     h_SumGPU[MAX_GPU_COUNT];//每个GPU的部分和

    float 	sumGPU;	//多个GPU累计的总和
    double 	sumCPU;	// CPU计算的总和
	double	diff;	//二者误差

	int gpuBase; // 数据指针 
    int i, j;

    const int  BLOCK_N = 32;  // Kernle 块数量
    const int THREAD_N = 256; // 每个块的线程数量
    const int  ACCUM_N = BLOCK_N * THREAD_N; // 每个Kernel 中的线程数量

    printf("Starting simpleMultiGPU\n");
	//获取GPU个数，可以是两块显卡，或者一块显卡上两个GPU核心；
    /* checkCudaErrors(hipGetDeviceCount(&GPU_N)); */
    /* printf("CUDA-capable device count: %i\n", GPU_N); */
	int GPU_N;  // GPU 个数
	int DATA_N = 1048576 ; // 1024*1024； 1M

	if(argc==3){
		GPU_N = atoi(argv[1]);	
		DATA_N = DATA_N * atoi(argv[2]);  // main参数以兆为单位
	}
	else{ // has no extra parameters
		printf("2 extra parameters needed!\n");
		printf("./mgpu gpu_num  data_num(M)\n");
		return 0;
	}
    printf("Generating input data...\n\n");
    //Subdividing input data across GPUs
    //Get data sizes for each GPU
    for (i = 0; i < GPU_N; i++)
    {
        plan[i].dataN = DATA_N / GPU_N;//lean: data number for single GPU;
    }// plan[0~31], GPU_N = 2,只使用plan数组中的前两个。

    //Take into account "odd" data sizes
    for (i = 0; i < DATA_N % GPU_N; i++)
    {
        plan[i].dataN++; //不能整除的情况，余数从0到GPU_N,分配给前几个GPU
    }

    //Assign data ranges to GPUs
    /* gpuBase = 0; */

    for (i = 0; i < GPU_N; i++)
    {
        plan[i].h_Sum = h_SumGPU + i;  //= & hSumGPU[i];  lean: h_Sum is a pointer .  
        /* gpuBase += plan[i].dataN; */
    }

    //Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i)); // 先选设备
        checkCudaErrors(hipStreamCreate(&plan[i].stream));
        //Allocate memory
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Data, plan[i].dataN * sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Sum, ACCUM_N * sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Sum_from_device, ACCUM_N * sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data, plan[i].dataN * sizeof(float)));

        for (j = 0; j < plan[i].dataN; j++)
        {
            plan[i].h_Data[j] = (float)rand() / (float)RAND_MAX; // 填入dataN个随机数据(注意，每个plan[i].dataN可能不一样大，因为余数)
        }
    }

    //Start timing and compute on GPU(s)
    printf("Computing with %d GPUs...\n", GPU_N);
    StartTimer();

    //Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++)
    {
        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Copy input data from CPU
        checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof(float), hipMemcpyHostToDevice, plan[i].stream));

        //Perform GPU computations
        reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);
        getLastCudaError("reduceKernel() execution failed.\n");

        //Read back GPU results
        checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream));
    }

    //Process GPU results
    for (i = 0; i < GPU_N; i++)
    {
        float sum;

        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Wait for all operations to finish
        hipStreamSynchronize(plan[i].stream);

        //Finalize GPU reduction for current subvector
        sum = 0;

        for (j = 0; j < ACCUM_N; j++)
        {
            sum += plan[i].h_Sum_from_device[j];
        }

        *(plan[i].h_Sum) = (float)sum;

        //Shut down this GPU
        checkCudaErrors(hipHostFree(plan[i].h_Sum_from_device));
        checkCudaErrors(hipFree(plan[i].d_Sum));
        checkCudaErrors(hipFree(plan[i].d_Data));
        checkCudaErrors(hipStreamDestroy(plan[i].stream));
    }

    sumGPU = 0;

    for (i = 0; i < GPU_N; i++)
    {
        sumGPU += h_SumGPU[i];
    }

    printf("%5s %10s %10s \n","GPU_N","data(M)","Time(ms)");
    printf("%5d %10d %10.3f\n", GPU_N, DATA_N/1024/1024, GetTimer());

    // Compute on Host CPU
    printf("Computing with Host CPU...\n\n");

    sumCPU = 0;

    for (i = 0; i < GPU_N; i++)
    {
        for (j = 0; j < plan[i].dataN; j++)
        {
            sumCPU += plan[i].h_Data[j];
        }
    }

    // Compare GPU and CPU results
    printf("Comparing GPU and Host CPU results...\n");
    diff = fabs(sumCPU - sumGPU) / fabs(sumCPU);
    printf("  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU);
    printf("  Relative difference: %E \n\n", diff);

    // Cleanup and shutdown
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipHostFree(plan[i].h_Data));
    }

    exit((diff < 1e-5) ? EXIT_SUCCESS : EXIT_FAILURE);
}
