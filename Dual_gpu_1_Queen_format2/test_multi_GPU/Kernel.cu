#include "hip/hip_runtime.h"
// 2018.11.10   // very slow average 200s
// every thread check for a Queen. not a pair[]
// 2018.12.29 
//copy form CheckOne_Datafile_compare/gpu_1_Queen
// 2019.4.15 add TGPU structor, modify get_conflicts() ;
#include "Kernel.h"
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>
//#define DEBUG 
#define BLOCK_SIZE 512 

typedef struct
{
    //Host-side input data
    unsigned *h_Data;
    //Partial sum for this GPU
    unsigned *h_Sum;

    //Device buffers
    unsigned *d_Data;
	unsigned *d_Sum;

    //Stream for asynchronous command execution
    hipStream_t stream;

} TGPUplan;
//----------------------------Kernel----------------------------------------
__global__ void Ker_Warm(){
	// empty body, just warmup GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Check_Combination (
		unsigned int *d_combination,			// 
		unsigned int combination_size,		// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid >= combination_size)   
		return;
	if(tid==0)
		d_result[0]=0;


	int curX=tid;
	int curY=d_combination[tid];
	//check every queen after cur;
	for(int iX=tid+1 ; iX <= combination_size-1 ; iX++){
		int iY=d_combination[iX];
		if(iY == curY || iX+iY == curX+ curY  ||  iY -iX == curY - curX) { // not a	Permutations, it is random numbers. 
			//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", curX,curY,iX,iY,tid);
			atomicAdd ((unsigned int *)&d_result[0],1);
			// break;   // get all conflicts
		}
	}

}// end of Kernel


//----------------------------CPU Interface----------------------------------------
void setDevice(int i)
{
	checkCudaErrors( hipSetDevice( i )  );    
}
int  getDevice()
{
	int id=-1;
	checkCudaErrors( hipGetDevice( &id )  );    
	return id;
}
void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

	Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
}


unsigned int get_conflicts(int * com, int N)
{

    //Solver config
    TGPUplan      plan[2];
	int GPU_N=0;
    //GPU reduction results
    printf("Starting simpleMultiGPU\n");
    checkCudaErrors(hipGetDeviceCount(&GPU_N));

    printf("CUDA-capable device count: %i\n", GPU_N);

    printf("Generating input data...\n\n");

    //Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
	int i=0;
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipStreamCreate(&plan[i].stream));
        //Allocate memory
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Data, N  * sizeof(unsigned)));
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Sum,  1  * sizeof(unsigned)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data, N * sizeof(unsigned)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Sum,  1 * sizeof(unsigned)));  // hipHostMalloc , not malloc()

		memcpy( plan[i].h_Data,  com, N * sizeof(unsigned));  // 普通内存数据转入页锁定内存；
    }

    //Start timing and compute on GPU(s)
    printf("Computing with %d GPUs...\n", GPU_N);
    StartTimer();

    //Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++)
    {
        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Copy input data from CPU
        checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data, N * sizeof(unsigned), hipMemcpyHostToDevice, plan[i].stream));

        //Perform GPU computations
		Ker_Check_Combination<<<   ( N + BLOCK_SIZE-1)/BLOCK_SIZE  ,  BLOCK_SIZE , 0 , plan[i].stream >>>
			( plan[i].d_Data ,N  ,  plan[i].d_Sum) ;  
        getLastCudaError("reduceKernel() execution failed.\n");

        //Read back GPU results
        checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum, plan[i].d_Sum, 1 *sizeof(unsigned), hipMemcpyDeviceToHost, plan[i].stream));
    }

    //Process GPU results
    for (i = 0; i < GPU_N; i++)
    {
        unsigned sum;

        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Wait for all operations to finish
        hipStreamSynchronize(plan[i].stream);

        //Shut down this GPU
        checkCudaErrors(hipHostFree(plan[i].h_Data));
        checkCudaErrors(hipFree(plan[i].d_Sum));
        checkCudaErrors(hipFree(plan[i].d_Data));
        checkCudaErrors(hipStreamDestroy(plan[i].stream));
    }


    printf("  GPU Processing time: %f (ms)\n\n", GetTimer());
	printf("%d  == %d \n ", plan[0].h_Sum[0],  plan[1].h_Sum[0]); 
	if( plan[0].h_Sum[0] ==  plan[1].h_Sum[0])
		printf("Check Pass");
	else
		printf("Check False");

	unsigned int conflicts = plan[0].h_Sum[0];


    // Cleanup and shutdown
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipHostFree(plan[i].h_Sum));
    }

		printf("run OK!\n");
		return conflicts;
}
