#include "hip/hip_runtime.h"
 /*
  * 2019.4.15 9:05 复制自 SDK simpleMultiGPU， 简化头文件，用来尝试双卡并行。
  * 2019.4.15 9:17 修改为整型数据
  */
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

//-------------------------------------------------------------------------------------------------------

__global__ void Ker_Warm(){
	// empty body, just warmup GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Check_Combination (
		int *d_combination,			// 
		int combination_size,		// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid >= combination_size)   
		return;
	if(tid==0)
		d_result[0]=0;


	int curX=tid;
	int curY=d_combination[tid];
	//check every queen after cur;
	for(int iX=tid+1 ; iX <= combination_size-1 ; iX++){
		int iY=d_combination[iX];
		if(iY == curY || iX+iY == curX+ curY  ||  iY -iX == curY - curX) { // not a	Permutations, it is random numbers. 
			//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", curX,curY,iX,iY,tid);
			atomicAdd ((unsigned int *)&d_result[0],1);
			// break;   // get all conflicts
		}
	}

}// end of Kernel


//----------------------------CPU Interface----------------------------------------
void setDevice(int i)
{
	checkCudaErrors( hipSetDevice( i )  );    
}
int  getDevice()
{
	int id=-1;
	checkCudaErrors( hipGetDevice( &id )  );    
	return id;
}
void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

	Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
}
//-------------------------------------------------------------------------------------------------------
typedef struct
{
    //Host-side input data
    int dataN;
    unsigned *h_Data;

    //Partial sum for this GPU
    unsigned *h_Sum;

    //Device buffers
    unsigned *d_Data;
	unsigned *d_Sum;

    //Stream for asynchronous command execution
    hipStream_t stream;

} TGPUplan;
////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    //Solver config
    TGPUplan      plan[2];

    //GPU reduction results
    printf("Starting simpleMultiGPU\n");
    checkCudaErrors(hipGetDeviceCount(&GPU_N));

    printf("CUDA-capable device count: %i\n", GPU_N);

    printf("Generating input data...\n\n");

    //Subdividing input data across GPUs
    //Get data sizes for each GPU
    for (i = 0; i < GPU_N; i++)
    {
        plan[i].dataN = DATA_N / GPU_N;//lean: data number for single GPU;
    }

    //Take into account "odd" data sizes
    for (i = 0; i < DATA_N % GPU_N; i++)
    {
        plan[i].dataN++;
    }

    //Assign data ranges to GPUs
    gpuBase = 0;

    for (i = 0; i < GPU_N; i++)
    {
        plan[i].h_Sum = h_SumGPU + i;  //= & hSumGPU[i];  lean: h_Sum is a pointer .  
        gpuBase += plan[i].dataN;
    }

    //Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipStreamCreate(&plan[i].stream));
        //Allocate memory
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Data, plan[i].dataN * sizeof(unsigned)));
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Sum, ACCUM_N * sizeof(unsigned)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data, plan[i].dataN * sizeof(unsigned)));

        for (j = 0; j < plan[i].dataN; j++)
        {
            plan[i].h_Data[j] = (unsigned)rand() / (unsigned)RAND_MAX;
        }
    }

    //Start timing and compute on GPU(s)
    printf("Computing with %d GPUs...\n", GPU_N);
    StartTimer();

    //Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++)
    {
        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Copy input data from CPU
        checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof(unsigned), hipMemcpyHostToDevice, plan[i].stream));

        //Perform GPU computations
        reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);
        getLastCudaError("reduceKernel() execution failed.\n");

        //Read back GPU results
        checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N *sizeof(unsigned), hipMemcpyDeviceToHost, plan[i].stream));
    }

    //Process GPU results
    for (i = 0; i < GPU_N; i++)
    {
        unsigned sum;

        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Wait for all operations to finish
        hipStreamSynchronize(plan[i].stream);

        //Finalize GPU reduction for current subvector
        sum = 0;

        for (j = 0; j < ACCUM_N; j++)
        {
            sum += plan[i].h_Sum_from_device[j];
        }

        *(plan[i].h_Sum) = (unsigned)sum;

        //Shut down this GPU
        checkCudaErrors(hipHostFree(plan[i].h_Sum_from_device));
        checkCudaErrors(hipFree(plan[i].d_Sum));
        checkCudaErrors(hipFree(plan[i].d_Data));
        checkCudaErrors(hipStreamDestroy(plan[i].stream));
    }

    sumGPU = 0;

    for (i = 0; i < GPU_N; i++)
    {
        sumGPU += h_SumGPU[i];
    }

    printf("  GPU Processing time: %f (ms)\n\n", GetTimer());

    // Compute on Host CPU
    printf("Computing with Host CPU...\n\n");

    sumCPU = 0;

    for (i = 0; i < GPU_N; i++)
    {
        for (j = 0; j < plan[i].dataN; j++)
        {
            sumCPU += plan[i].h_Data[j];
        }
    }

    // Compare GPU and CPU results
    printf("Comparing GPU and Host CPU results...\n");
    diff = fabs(sumCPU - sumGPU) / fabs(sumCPU);
    printf("  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU);
    printf("  Relative difference: %E \n\n", diff);

    // Cleanup and shutdown
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipHostFree(plan[i].h_Data));
    }

    exit((diff < 1e-5) ? EXIT_SUCCESS : EXIT_FAILURE);
}
