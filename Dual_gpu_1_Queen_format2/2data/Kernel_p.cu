#include "hip/hip_runtime.h"
// 2018.11.10   // very slow average 200s
// every thread check for a Queen. not a pair[]
// 2018.12.29 
//copy form CheckOne_Datafile_compare/gpu_1_Queen
#include "Kernel_p.h"
#define DEBUG 
#define BLOCK_SIZE 512 

//----------------------------Kernel----------------------------------------
__global__ void Ker_Warm(){
	// empty body, just warmup GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Check_Combination3 (
		unsigned int *d_combination,			// 
		unsigned int combination_size,		// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{

	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	d_result[0] =0;
	if( tid == 0 ) {
		for(int i =0 ;i<combination_size ; i++){
			printf("%4d  ", d_combination[i]);
			d_result[0] += d_combination[i];
		}

		printf("\nsum = %8d \n ", d_result[0]);
	}
}

__global__ void Ker_Check_Combination (
		unsigned int *d_combination,			// 
		unsigned int combination_size,		// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid >= combination_size)   
		return;
	if(tid==0)
		d_result[0]=0;


	int curX=tid;
	int curY=d_combination[tid];
	//check every queen after cur;
	for(int iX=tid+1 ; iX <= combination_size-1 ; iX++){
		int iY=d_combination[iX];
		if(iY == curY || iX+iY == curX+ curY  ||  iY -iX == curY - curX) { // not a	Permutations, it is random numbers. 
			//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", curX,curY,iX,iY,tid);
			atomicAdd ((unsigned int *)&d_result[0],1);
			// break;   // get all conflicts
		}
	}

}// end of Kernel


//----------------------------CPU Interface----------------------------------------
void setDevice(int i)
{
	checkCudaErrors( hipSetDevice( i )  );    
}
int  getDevice()
{
	int id=-1;
	checkCudaErrors( hipGetDevice( &id )  );    
	return id;
}
void  warmGPU()  // warm a single GPU
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

	Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "Worm launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
}
void  warmGPU0_1()  // warm GPU 0 and 1 in diffirent streams// wrong??
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

    hipStream_t stream0;
    hipStream_t stream1;

	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipStreamCreate(&stream0));
	Ker_Warm <<<1,1,0 , stream0>>>	();
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors(hipStreamCreate(&stream1));
	Ker_Warm <<<1,1,0 , stream1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "Worm launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipStreamDestroy(stream0));
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors(hipStreamDestroy(stream1));
}


void get_conflicts(unsigned int * combi0,  unsigned int * combi1, unsigned int size, unsigned *res0, unsigned *res1)
{
	/* printf("\n----get_conflicts() begin! \n"); */

	unsigned *h_combi0= combi0 ; // 不是页锁定内存，是主程序直接传入的数组
	unsigned *h_combi1= combi1 ; // 不是页锁定内存，是主程序直接传入的数组

	unsigned *d_combi0= 0;
	unsigned *d_combi1= 0;
	unsigned * h_result0 = 0;
	unsigned * h_result1 = 0;
	unsigned * d_result0 = 0;
	unsigned * d_result1 = 0;
	hipError_t cuda_err; 
    hipStream_t stream0;
    hipStream_t stream1;

	//--------------cpu----------------------------------
    //int  GPU_N;
    //checkCudaErrors(hipGetDeviceCount(&GPU_N));
	cuda_err = hipSuccess;
	//---------------GPU0 memroy alloc-----------------------------
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipStreamCreate(&stream0));

	checkCudaErrors(hipMalloc((void **)&d_combi0, 		size * sizeof( unsigned)));
	checkCudaErrors(hipMalloc((void **)&d_result0 ,     1	 * sizeof( unsigned)));
	checkCudaErrors(hipHostMalloc((void **)&h_result0,  1 	 * sizeof(unsigned)));  // hipHostMalloc , not malloc()  
	/* memcpy( h_combi0,  combi , size  * sizeof(unsigned));  */

	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "GPU 0 alloc d_combi error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//---------------GPU1 memory alloc-----------------------------
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors(hipStreamCreate(&stream1));
	checkCudaErrors(hipMalloc((void **)&d_combi1, 		size * sizeof( unsigned)));
	checkCudaErrors(hipMalloc((void **)&d_result1 ,     1   * sizeof( unsigned)));
	checkCudaErrors(hipHostMalloc((void **)&h_result1,  1   * sizeof( unsigned)));  // hipHostMalloc , not malloc()  
	/* memcpy( h_combi1,  combi , size * sizeof(unsigned));  */

	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "GPU1  alloc d_combi error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  

// ansync transfer, run Kernel , and transfer result back;
// -----------------GPU0----------------------------------------------------------------------------
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMemcpyAsync(d_combi0, h_combi0, size * sizeof( unsigned), hipMemcpyHostToDevice, stream0));

	int GRID_SIZE  = ( size + BLOCK_SIZE-1)/BLOCK_SIZE ;
	/* printf("gridsize = %d, blocksize =%d, queens = %d\n", GRID_SIZE, BLOCK_SIZE, size ); */
	checkCudaErrors(hipSetDevice(0));
	/* printf( "Switch to device : %d\n", getDevice()); */

	Ker_Check_Combination<<<   GRID_SIZE,  BLOCK_SIZE , 0, stream0 >>>
		(d_combi0 ,size ,  d_result0);  

	getLastCudaError("Kernel() in divece 0 execution failed.\n");
	checkCudaErrors(hipMemcpyAsync(h_result0, d_result0, 1 * sizeof(unsigned), hipMemcpyDeviceToHost,stream0));
// -----------------GPU1----------------------------------------------------------------------------
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors(hipMemcpyAsync(d_combi1, h_combi1, size * sizeof( unsigned), hipMemcpyHostToDevice, stream1));
	/* printf( "Switch to device : %d\n", getDevice()); */
	cuda_err= hipSuccess;
	Ker_Check_Combination<<<   ( size + BLOCK_SIZE-1)/BLOCK_SIZE  ,  BLOCK_SIZE , 0, stream1 >>>
		(d_combi1 ,size ,  d_result1);  

	getLastCudaError("Kernel() in divece 1 execution failed.\n");
	checkCudaErrors(hipMemcpyAsync(h_result1, d_result1, 1 * sizeof(unsigned), hipMemcpyDeviceToHost,stream1));
//------------------GPU0 同步-----------------------------------------------------------
	checkCudaErrors(hipSetDevice(0));
	hipStreamSynchronize(stream0);

//------------------GPU1 同步-----------------------------------------------------------
	checkCudaErrors(hipSetDevice(1));
	hipStreamSynchronize(stream1);

// ----------------------CPU ---------------------------------
	 *res0= h_result0[0];
	 *res1= h_result1[0];
	//************************************************************************************************************

	//--------GPU0---------------------------
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors( hipFree(d_combi0) );
	checkCudaErrors( hipFree(d_result0)      );
	/* checkCudaErrors(hipHostFree(h_combi0)); */
	checkCudaErrors(hipHostFree(h_result0)); 
	checkCudaErrors(hipStreamDestroy(stream0));
	//--------GPU0---------------------------
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors( hipFree(d_combi1) );
	checkCudaErrors( hipFree(d_result1)      );
	/* checkCudaErrors(hipHostFree(h_combi1)); */
	checkCudaErrors(hipHostFree(h_result1)); 
	checkCudaErrors(hipStreamDestroy(stream1));

	/* printf(" get_conflicts()  run OK !\n"); */

}
