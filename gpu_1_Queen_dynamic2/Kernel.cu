#include "hip/hip_runtime.h"
/* 2018.11.10   // very slow average 200s
* 	every thread check for a Queen. not a pair[]
* 2018.12.4
*	use dynamic parallel to speed up large N number.
*   in Sub Kernel. set x of header Queen = -1 to avoid caculate global index of subArr[].
*/

#include "Kernel.h"
//#define DEBUG 		0
//every block has x Queen, i.e.  x raws.
#define BLOCK_SIZE 	512 

// number that every thread int sub_Kernel loops ,i.e. pairs that this thread checked;
// threshold
#define SUB_TASK  (128*1024)
// thread numbers in every sub block 
#define SUB_BLOCK_SIZE  32

//----------------------------Kernel----------------------------------------
__global__ void Ker_Warm(){
	// empty body, just warmup GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Sub(
		int super_tid,
		/*int X,*/     // use -1 instead.
		int Y,
		int *SubArr,	// Arr[super.tid+1, N-1]
		int SubLen ,    // N-1 -( super.tid+1) + 1
		unsigned int *d_result           )
{
	const int sub_tid = blockDim.x * blockIdx.x + threadIdx.x;   	// [0.. 127]
	//int my_job_begin = SUB_TASK * sub_tid ;
	//int my_job_end	 = SUB_TASK * (sub_tid+1);
	int X=-1;  // convert sub[] index to global[] index;
	for(int sx = SUB_TASK * sub_tid;  sx< SUB_TASK * (sub_tid + 1) && sx < SubLen ; sx++){
		int sy = SubArr[sx];  // Arr[super_tid + 1 + sx]
		# ifdef DEBUG
		printf("Super [%5d] Sub[%5d]-------->>>compare %5d to %5d\n ", super_tid, sub_tid, X, sx );
		#endif
		if(Y == sy || X+Y == sx + sy  ||  Y -X == sy - sx) { // not a	Permutations, it is random numbers. 
			atomicAdd ((unsigned int *)&d_result[0],1);
		}//if
	}//for

}

// check for a Queen , i.e. a round.  compare arr[tid] to arr[tid+1 .. N-1].
__global__ void Ker_Check_Combination (
		int *Arr,			// arr[0,N-1]
		int N,						// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   
	if(tid >= N -1)  // last queen arr[N-1] need't check.   
		return;

	
	//int curX=tid;  // curX is global index in Arr[], in subArr, use -1
	int curY=Arr[tid];
	int len_of_rest_Queens = N -1-tid;
	# ifdef DEBUG
	printf("Super[%5d]-------------------->>>  len_of_rest_Queens %5d, sub_ker_threads %5d  \n", 
										 tid , len_of_rest_Queens    , sub_ker_threads);
	#endif

	if( len_of_rest_Queens > SUB_TASK ) {
		//call sub kernel. 
		//printf("Sub Kernel called in thread:%5d\n", tid);
		int sub_ker_threads = (len_of_rest_Queens + SUB_TASK -1) / SUB_TASK ;
		Ker_Sub<<< ( sub_ker_threads + SUB_BLOCK_SIZE -1 ) / SUB_BLOCK_SIZE , SUB_BLOCK_SIZE >>> ( tid , /*curX,  -1 instead */  curY, &Arr[tid+1] ,  len_of_rest_Queens, d_result);
	}
	else {
		// for small tasks, check by itself.
		int curX=tid;
		//check every queen after cur;
		for(int iX=tid+1 ; iX <= N-1 ; iX++){
			int iY=Arr[iX];
			if(iY == curY || iX+iY == curX+ curY  ||  iY -iX == curY - curX) { // not a	Permutations, it is random numbers. 
				//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", curX,curY,iX,iY,tid);
				atomicAdd ((unsigned int *)&d_result[0],1);
				// break;   // get all conflicts
			}
		}

	}



}// end of Kernel


//----------------------------CPU Interface----------------------------------------
void setDevice(int i)
{
	checkCudaErrors( hipSetDevice( i )  );    
}
void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

	Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
}


void show_config(int combination_size)
{
	printf("%20s\t%20s\t%20s\n","super block size","sub block size","sub task size");
	printf("%20d\t%20d\t%20d\n",BLOCK_SIZE, SUB_BLOCK_SIZE, SUB_TASK);
	printf("Ker_Check_Combination<<< %5d,%5d >>> \n", (combination_size + BLOCK_SIZE-1)/BLOCK_SIZE  ,  BLOCK_SIZE );
}

unsigned int get_conflicts(int * combination, int combination_size)
{
	int *h_combination = 0;  //store a number in [1~N]
	int *d_combination = 0;
	unsigned int * h_result= 0;
	unsigned int * d_result= 0;
	// timer 
	//std::chrono::time_point<std::chrono::system_clock> c11_start, c11_end;
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//float cuda_time=0;
	//int cpu_time=0; 
	//cuda status var;
	hipError_t cuda_err; 
	// Allocate CPU memory and initialize data.

	// init h_combination & timer.
	//c11_start =std::chrono::system_clock::now();
	//h_combination   =(int *)malloc(combination_size   * sizeof(int));  // need not allocate & free memory
	h_result        =(unsigned int *)malloc(	1                * sizeof(unsigned int));

	if(h_result==NULL )
	{
		printf("malloc h_result  error \ni");
		exit(1);
	}

	h_combination= combination;

	cuda_err = hipSuccess;
	//hipEventRecord(start, 0);
	// allocate GPU mem
	checkCudaErrors(hipMalloc((void **)&d_combination, combination_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_result ,     1                * sizeof(unsigned int)));
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "alloc d_combination error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//else
	//	fprintf(stderr, "alloc d_combination  successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","GPU mem allocate  time=",cuda_time);
	//************************************************************************************************************


	//combination  H->D	
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(d_combination, h_combination, combination_size * sizeof(int), hipMemcpyHostToDevice));
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","combination[] tranfer time =", cuda_time);

	// Execute  & timer
	//hipEventRecord(start, 0);

	Ker_Check_Combination<<<   ( combination_size + BLOCK_SIZE-1)/BLOCK_SIZE  ,  BLOCK_SIZE  >>>
			(d_combination ,combination_size ,  d_result);  

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	# ifdef DEBUG
	else
		fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif

	//checkCudaErrors(hipDeviceSynchronize());

	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","CUDA Kernel run time=",cuda_time);


	// D->H  and timer	
	h_result[0]=0;
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(h_result, d_result, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost));
	cuda_err = hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "D->H error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	# ifdef DEBUG
	else
		fprintf(stderr, "D->H successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","CUDA D->H time =  ", cuda_time);

			//printf("testORI Kernel OK!  result= %d \n",h_result[0]);
	unsigned int conflicts = h_result[0];
	//************************************************************************************************************

	//free memory
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	// free(h_combination); //it is a pointer point to static array a[][], need not to free;
	free(h_result);
	checkCudaErrors( hipFree(d_combination) );
	checkCudaErrors( hipFree(d_result)      );
	checkCudaErrors( hipDeviceSynchronize() );
	//checkCudaErrors(  hipDeviceReset()     );
	return  conflicts;
}
