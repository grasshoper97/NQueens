#include "hip/hip_runtime.h"
// 2018.11.10   // very slow average 200s
// every thread check for a Queen. not a pair[]
// 2018.12.29 
//copy form CheckOne_Datafile_compare/gpu_1_Queen
#include "Kernel.h"
//#define DEBUG 
#define BLOCK_SIZE 512 

//----------------------------Kernel----------------------------------------
__global__ void Ker_Warm(){
	// empty body, just warmup GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Check_Combination (
		int *d_combination,			// 
		int combination_size,		// length of combinations =queen number =N
		unsigned int *d_result				// return conflicts count.
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid >= combination_size)   
		return;
	if(tid==0)
		d_result[0]=0;


	int curX=tid;
	int curY=d_combination[tid];
	//check every queen after cur;
	for(int iX=tid+1 ; iX <= combination_size-1 ; iX++){
		int iY=d_combination[iX];
		if(iY == curY || iX+iY == curX+ curY  ||  iY -iX == curY - curX) { // not a	Permutations, it is random numbers. 
			//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", curX,curY,iX,iY,tid);
			atomicAdd ((unsigned int *)&d_result[0],1);
			// break;   // get all conflicts
		}
	}

}// end of Kernel


//----------------------------CPU Interface----------------------------------------
void setDevice(int i)
{
	checkCudaErrors( hipSetDevice( i )  );    
}
int  getDevice()
{
	int id=-1;
	checkCudaErrors( hipGetDevice( &id )  );    
	return id;
}
void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;
	hipEventRecord(start, 0);

	Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	//checkCudaErrors(  hipDeviceSynchronize()  );

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","warmup() run time=",cuda_time);
	#endif
}


unsigned int get_conflicts(int * combination, int combination_size)
{
	// 预热GPU
	//warmup<<<1, 1>>>();
	// Create input data
	int *h_combination = 0;  //store a number in [1~N]
	int *d_combination = 0;
	unsigned int * h_result = 0;
	unsigned int * d_result = 0;
	// timer 
	//std::chrono::time_point<std::chrono::system_clock> c11_start, c11_end;
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//float cuda_time=0;
	//int cpu_time=0; 
	//cuda status var;
	hipError_t cuda_err; 
	// Allocate CPU memory and initialize data.

	// init h_combination & timer.
	//c11_start =std::chrono::system_clock::now();
	//h_combination   =(int *)malloc(combination_size   * sizeof(int));  // need not allocate & free memory
	h_result        =(unsigned int *)malloc(	1                * sizeof(unsigned int));

	if(h_result==NULL )
	{
		printf("malloc h_result  error \ni");
		exit(1);
	}

	h_combination= combination;

	cuda_err = hipSuccess;
	//hipEventRecord(start, 0);
	// allocate GPU mem
	checkCudaErrors(hipMalloc((void **)&d_combination, combination_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_result ,     1                * sizeof(int)));
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "alloc d_combination error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//else
	//	fprintf(stderr, "alloc d_combination  successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","GPU mem allocate  time=",cuda_time);
	//************************************************************************************************************


	//combination  H->D	
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(d_combination, h_combination, combination_size * sizeof(int), hipMemcpyHostToDevice));
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","combination[] tranfer time =", cuda_time);

	// Execute  & timer
	//hipEventRecord(start, 0);
	Ker_Check_Combination<<<   ( combination_size + BLOCK_SIZE-1)/BLOCK_SIZE  ,  BLOCK_SIZE  >>>
			(d_combination ,combination_size ,  d_result);  

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
	#ifdef DEBUG
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
		exit(EXIT_FAILURE);
	}  
	else
	#ifdef DEBUG
			fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif

	//checkCudaErrors(hipDeviceSynchronize());

	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","CUDA Kernel run time=",cuda_time);


	// D->H  and timer	
	h_result[0]=0;
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(h_result, d_result, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost));
	cuda_err = hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
	#ifdef DEBUG
		fprintf(stderr, "D->H error! (error code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
		exit(EXIT_FAILURE);
	}  
	#ifdef DEBUG
	else
			fprintf(stderr, "D->H successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&cuda_time, start, stop);
	//if(DEBUG) printf("%-40s %f ms \n","CUDA D->H time =  ", cuda_time);

			//printf("testORI Kernel OK!  result= %d \n",h_result[0]);
	unsigned int conflicts = h_result[0];
	//************************************************************************************************************

	//free memory
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	// free(h_combination); //it is a pointer point to static array a[][], need not to free;
	free(h_result);
	checkCudaErrors( hipFree(d_combination) );
	checkCudaErrors( hipFree(d_result)      );
	checkCudaErrors( hipDeviceSynchronize() );
	//checkCudaErrors(  hipDeviceReset()     );
	return  conflicts;
}
