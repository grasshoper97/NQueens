#include "hip/hip_runtime.h"
/*
* 2018.12.3  
* add global vars d_pairs_i, d_pairs_j, pair_size 
* h_result/d_result/reaturn type change to unsigned int  
* 2018.12.4  assign d_result =0 use memcpy. not in Ker.
*/
#include "Kernel.h"
#define DEBUG 0
#define BLOCK_SIZE 512 

// global var;
int pairs_size=0;
int *d_pairs_i=0;
int *d_pairs_j=0;

//..................................................................
__global__ void Ker_Warm(){
	// empty body, just Ker_Warm GPU;
	if(threadIdx.x == 0 )
		printf("GPU is OK!\n");
}

__global__ void Ker_Check_Combination (
		int *d_combination,		// a combination to be checkeda, store in d_com[1~N], d_com[0] is unused, for caculate gradient esaily.
		int combination_size,	// length of combinations =queen number =N
		int *I, 				// pair X   , store in I[0 ~~ pairs_size-1]
		int *J,					// pari Y   , store in I[0 ~~ pairs_size-1]
		int pairs_size,			// pair number , =N*(N-1)/2
		unsigned int *d_result
		)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;   	// use shared mem,so must be in a Block, need not global thread No.
	//printf("block =%d, thread=%d, tid=%d\n",blockIdx.x, threadIdx.x ,tid);
	if(tid>=pairs_size)  // pairs_size = N*(N-1)/2
		return;


	// N*(N-1)/2 threads check pair
	int Xi=I[tid];   // tid in [0 ~~ pairs_size]
	int Xj=J[tid];   // I > J
	int Yi=d_combination[Xi];  // i,j in [1 ~~ N]
	int Yj=d_combination[Xj];
	if( Yi == Yj || Xi+Yi == Xj+Yj  ||  Yi -Xi == Yj - Xj) { //use +- instead of *,  can judge random sequence 
		//printf("-------->>> (%5d,%5d) (%5d,%5d) thread:%5d  \n ", Xj,Yj,Xi,Yi,tid);
		atomicAdd ((unsigned int *)&d_result[0],1);
		//printf("add 1: block =%d, thread=%d, tid=%d pair (%2d,%2d)\n",blockIdx.x, threadIdx.x ,tid, Xi, Xj);
	}


}// end of Kernel

//............................ cpu Interface......................................

void gen_pairs(int * h_pairs_i, int * h_pairs_j, int len, int n)
{
	int index=0; 
  //check every queen after current queen.
	for (int pi =0; pi <=n-2; pi++){         //generate check pairs   [ 1~N-1,   i+1~N ], store in h_pair[0~ len-1]
		for(int pj =pi+1 ; pj<=n-1 ; pj++){ 
			h_pairs_i[index]=pi; 
			h_pairs_j[index]=pj; 
			index++; 
		} 
	}//end  
	//if(index == len)
		//printf("index == len \n" );
}


// run only once, can be use multi times
void gen_transfer_pairs( int N)
{
	int *h_pairs_i=0; // after data trans to GPU, this memory is free.
	int *h_pairs_j=0;
	//N is combination_size;
	pairs_size=N*(N-1)/2;   //glabal vars
	h_pairs_i       =(int *)malloc(pairs_size         * sizeof(int));
	h_pairs_j       =(int *)malloc(pairs_size         * sizeof(int));
	if( h_pairs_i==NULL || h_pairs_j==NULL  )
	{
		printf("malloc h_pairs  error \n");
		exit(1);
	}
	// gen pairs[] in CPU
	gen_pairs(h_pairs_i, h_pairs_j, pairs_size, N);

	// alloc GPU mem
	hipError_t cuda_err; 
	cuda_err = hipSuccess;
	checkCudaErrors(hipMalloc((void **)&d_pairs_i,     pairs_size       * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_pairs_j,     pairs_size       * sizeof(int)));
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "alloc d_pair[] error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//else
		//fprintf(stderr, "alloc d_pair[] successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	
	//H to D
	checkCudaErrors(hipMemcpy(d_pairs_i, h_pairs_i, pairs_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_pairs_j, h_pairs_j, pairs_size * sizeof(int), hipMemcpyHostToDevice));
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "transfer  d_pair[] error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//else
		//fprintf(stderr, "transfer d_pair[] successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	free(h_pairs_i);
	free(h_pairs_j);
}

void release_d_pairs()
{
	checkCudaErrors(hipFree(d_pairs_i));
	checkCudaErrors(hipFree(d_pairs_j));
}

void  warmGPU() 
{
	hipError_t cuda_err; 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float cuda_time=0;

	hipEventRecord(start, 0);
		Ker_Warm <<<1,1>>>	();

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
	{	
	#ifdef DEBUG
	fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));
	#endif
	}
	checkCudaErrors(hipDeviceSynchronize());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cuda_time, start, stop);
	#ifdef DEBUG
	printf("%-40s %f ms \n","Ker_Warm() run time=",cuda_time);
	#endif
}

unsigned int get_conflicts(int * combination, int combination_size)
{
	int *h_combination = 0;  //store a number in [1~N]
	int *d_combination = 0;
	unsigned int *h_result= 0;
	unsigned int *d_result= 0;  // for automic opration
	hipError_t cuda_err; 
	h_result        =(unsigned int *)malloc(	1                * sizeof(unsigned int));

	if(  h_result==NULL )
	{
		printf("malloc h_result  error \ni");
		exit(1);
	}

	h_result[0]=0;
	h_combination= combination;

	cuda_err = hipSuccess;
	// allocate GPU mem
	checkCudaErrors(hipMalloc((void **)&d_combination, combination_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)&d_result ,     1                * sizeof(unsigned int)));
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		fprintf(stderr, "alloc d_combination error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	//combination  H->D	
	checkCudaErrors(hipMemcpy(d_combination, h_combination, combination_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_result		, h_result	   , 1				  * sizeof(unsigned int), hipMemcpyHostToDevice));

	//set d_pairs[]
	gen_transfer_pairs( combination_size );

	//call Kernel<<< >>>
	Ker_Check_Combination<<<   (pairs_size + BLOCK_SIZE-1)/BLOCK_SIZE ,  BLOCK_SIZE  >>>
				(d_combination ,combination_size  ,d_pairs_i, d_pairs_j, pairs_size, d_result);  

	cuda_err= hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{   
		if(DEBUG) fprintf(stderr, "Failed to launch (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
		if(DEBUG) fprintf(stderr, "launch successed! ( code= %s)!\n", hipGetErrorString(cuda_err));


	// D->H  and timer	
	//hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(h_result, d_result, 1 * sizeof(unsigned int), hipMemcpyDeviceToHost)); // wait for Kernel finished.
	cuda_err = hipSuccess;
	cuda_err = hipGetLastError();    
	if (cuda_err != hipSuccess) 
	{                      
		if(DEBUG) fprintf(stderr, "D->H error! (error code= %s)!\n", hipGetErrorString(cuda_err));
		exit(EXIT_FAILURE);
	}  
	else
		if(DEBUG) fprintf(stderr, "D->H successed ! ( code= %s)!\n", hipGetErrorString(cuda_err));
	unsigned int conflicts = h_result[0];
	//************************************************************************************************************

	//free memory
	free(h_result);
	checkCudaErrors(hipFree(d_combination));
	checkCudaErrors(hipFree(d_result));
	release_d_pairs();

	return  conflicts;
}
